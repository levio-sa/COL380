#include <string>
#include <fstream>
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;


pair<int,int> read_img(string img_file, int *&img){
    ifstream ifs(img_file);

    while(ifs.is_open()){
        int x,y,tmp;
        ifs >> x >> y;

        hipMallocManaged(&img, x*y*3*sizeof(int));

        for(int i = 0; i < x; ++i){
            for(int j = 0; j < y; ++j){
                for(int k = 0; k < 3; ++k){
                    ifs>>tmp;
                    // cout<<tmp<<"\n";
                    img[(x-1-i)*y*3 +j*3 +k] = tmp;
                    // cout<<img[(x-1-i)*y*3 +j*3 +k];
                }
            }
        }
        // cout<<"HELLO"<<endl;
        ifs.close();
        return {x,y};
    }

}

__device__ float square_root(float x){
    return sqrt(x);
}

__device__ void bin_inter(int* &data_img, int n, int m, float x, float y, float* res){
    int a = x/1;
    int b = y/1;
    x-=a;
    y-=b;
    res = new float[3];

    res[0] = data_img[a*m*3+b*3+0]*(1-x)*(1-y)+data_img[(a+1)*m*3+b*3+0]*(x)*(1-y)+data_img[a*m*3+(b+1)*3+0]*(1-x)*(y)+data_img[(a+1)*m*3+(b+1)*3+0]*(x)*(y);
    res[1] = data_img[a*m*3+b*3+1]*(1-x)*(1-y)+data_img[(a+1)*m*3+b*3+1]*(x)*(1-y)+data_img[a*m*3+(b+1)*3+1]*(1-x)*(y)+data_img[(a+1)*m*3+(b+1)*3+1]*(x)*(y);
    res[2] = data_img[a*m*3+b*3+2]*(1-x)*(1-y)+data_img[(a+1)*m*3+b*3+2]*(x)*(1-y)+data_img[a*m*3+(b+1)*3+2]*(1-x)*(y)+data_img[(a+1)*m*3+(b+1)*3+2]*(x)*(y);

    // return res;
}


__global__ void main_func(int *data_img, int n, int m, int *q_img, int a, int b, float* rmsd, float q_avg, float th1, float th2){

    // int index = blockIdx.x*blockDim.x + threadIdx.x;
    // int i = index/m;
    // int j = index%m;
    int i = blockIdx.x;
    int j = blockIdx.y;

    // printf("helo\n");

    float avg1 = 0;
    float avg2 = 0;
    float avg3 = 0;

    if(i==840 && j==900){
        printf("helo\n");
    }

    // 0
    if(i+a<n && j+b<m){
        avg1 = 0;
        for(int x=i; x<i+a; ++x){
            for(int y=j; y<j+b; ++y){
                avg1 += ((float)(data_img[x*m*3+y*3+0]+data_img[x*m*3+y*3+1]+data_img[x*m*3+y*3+2]));
            }
        }
        avg1 = avg1/((float)(3*a*b));
        if(i==840 && j==900){
            printf("%f\n",abs(q_avg-avg1));
        }
        // printf("%f\n",abs(q_avg-avg1));
        if(abs(q_avg-avg1)<th2){
            if(i==840 && j==900){
                printf("%f\n",abs(q_avg-avg1));
            }
            // printf("%f\n",abs(q_avg-avg1));
            float rms1 = 0;
            for(int x=i;x<i+a;++x){
                for(int y=j;y<j+b;++y){
                    for(int z=0;z<3;++z){
                        float t = data_img[x*m*3+y*3+z]-q_img[(x-i)*b*3+(y-j)*3+z];
                        // rms1+=pow((data_img[x*m*3+y*3+z]-q_img[(x-i)*b*3+(y-j)*3+z]),2);
                        rms1+=t*t;
                    }
                }
            }
            rms1 = rms1/((float)a*b*3);
            rms1 = square_root(rms1);
            if(i==840 && j==900){
                printf("%f\n",abs(rms1));
            }
            if(rms1<th1){
                // printf("%f\n",rms1);
                rmsd[i*m*3+j*3] = (rms1);
                // printf("%f\n",rms1);
            }
            else{
                // avg1 = -1;
                rmsd[i*m*3+j*3] = -1;
            }
        } 
        else{
            // avg1 = -1;
            rmsd[i*m*3+j*3] = -1;
        }       
    }
    else{
        // avg1 = -1;
        rmsd[i*m*3+j*3] = -1;
    }

    // +45
    float xoff = (float)a/(float)square_root(2);
    float yoff = (float)b/(float)square_root(2);
    int x1 = i;
    int x2 = (float)i + ((float)(a+b))/((float)square_root(2)) + (float)1;
    int y1 = (float)j - (float)a/(float)square_root(2);
    int y2 = (float)j + (float)b/(float)square_root(2) + (float)1;
    if(y1>=0 && y2<m && x2<n){
        avg2 = 0;
        for(int x=x1;x<x2;++x){
            for(int y=y1;y<y2;++y){
                avg2 += ((float)(data_img[x*m*3+y*3+0]+data_img[x*m*3+y*3+1]+data_img[x*m*3+y*3+2]))/((float)3);
            }
        }
        avg2 = avg2/(float)(4*xoff*yoff);
        if(abs(q_avg-avg2)<th2){
            float rms2 = 0.0;
            for(int x=0;x<a;++x){
                for(int y=0;y<b;++y){
                    float p1,p2;
                    p2 = (float)j+((float)x/(float)square_root(2))-((float)y/(float)square_root(2));
                    p1 = (float)i+((float)x/(float)square_root(2))+((float)y/(float)square_root(2));
                    float* temp1;
                    bin_inter(data_img, n,m,p1,p2, temp1);
                    for(int z=0;z<3;++z){
                        rms2+=pow((temp1[z]-q_img[x*b*3+y*3+z]),2);
                    }
                }
            }
            rms2 = rms2/((float)a*b*3);
            rms2 = square_root(rms2);
            if(rms2<th1){
                rmsd[i*m*3+j*3+1] = (rms2);
                // printf("%f\n",rms2);
            }
            else{
                // avg1 = -1;
                rmsd[i*m*3+j*3+1] = -1;
            }
        } 
        else{
            // avg[i*m*3+j*3+1] = -1;
            rmsd[i*m*3+j*3+1] = -1;
        }
    }
    else{
        // avg[i*m*3+j*3+1] = -1;
        rmsd[i*m*3+j*3+1] = -1;
    }

    // -45
    x1 = (float)i - (float)b/(float)square_root(2);
    x2 = (float)i + (float)a/(float)square_root(2) + (float)1;
    y1 = j;
    y2 = (float)j + ((float)(a+b))/((float)square_root(2)) + (float)1;
    if(x1>=0 && y2<m && x2<n){
        avg3 = 0;
        for(int x=x1;x<x2;++x){
            for(int y=y1;y<y2;++y){
                avg3 += ((float)(data_img[x*m*3+y*3+0]+data_img[x*m*3+y*3+1]+data_img[x*m*3+y*3+2]))/((float)3);
            }
        }
        avg3 = avg3/(float)(4*xoff*yoff);
        if(abs(q_avg-avg3)<th2){
            float rms3 = 0.0;
            for(int x=0;x<a;++x){
                for(int y=0;y<b;++y){
                    float p1,p2;
                    p2 = (float)j+((float)x/(float)square_root(2))+((float)y/(float)square_root(2));
                    p1 = (float)i+((float)x/(float)square_root(2))-((float)y/(float)square_root(2));
                    float* temp1;
                    bin_inter(data_img, n,m,p1,p2, temp1);
                    for(int z=0;z<3;++z){
                        rms3+=pow((temp1[z]-q_img[x*b*3+y*3+z]),2);
                    }
                }
            }
            rms3 = rms3/((float)a*b*3);
            rms3 = square_root(rms3);
            if(rms3<th1){
                rmsd[i*m*3+j*3+2] = (rms3);
            }
            else{
                // avg1 = -1;
                rmsd[i*m*3+j*3+2] = -1;
            }
        } 
        else{
            // avg[i*m*3+j*3+1] = -1;
            rmsd[i*m*3+j*3+2] = -1;
        }
    }
    else{
        // avg[i*m*3+j*3+1] = -1;
        rmsd[i*m*3+j*3+2] = -1;
    }

    // printf("%f\n",rmsd[i*m*3+j*3+0]);
    // printf("%f\n",rmsd[i*m*3+j*3+1]);
    // printf("%f\n",rmsd[i*m*3+j*3+2]);

}


int main(int argc, char* argv[]){

    if(argc < 6){
        cout << "Invalid arguments"<<endl;
        exit(-1);
    }

    string data_img_file = argv[1];
    string q_img_file = argv[2];
    float th1 = stof(argv[3]);
    float th2 = stof(argv[4]);
    int n = stoi(argv[5]);

    
    int *data_img, *q_img;
    int r1,r2,r3,r4;

    pair<int,int> dim1, dim2;
    
    ifstream ifs(data_img_file);

    while(ifs.is_open()){
        int tmp;
        ifs >> r1 >> r2;

        hipMallocManaged(&data_img, r1*r2*3*sizeof(int));

        for(int i = 0; i < r1; ++i){
            for(int j = 0; j < r2; ++j){
                for(int k = 0; k < 3; ++k){
                    ifs>>tmp;
                    // cout<<tmp<<"\n";
                    data_img[(r1-1-i)*r2*3 +j*3 +k] = tmp;
                    // cout<<img[(x-1-i)*y*3 +j*3 +k];
                }
            }
        }
        // cout<<"HELLO"<<endl;
        ifs.close();
    }
    dim1 = {r1,r2};

    ifstream ifs1(q_img_file);

    while(ifs1.is_open()){
        int tmp;
        ifs1 >> r3 >> r4;

        hipMallocManaged(&q_img, r3*r4*3*sizeof(int));

        for(int i = 0; i < r3; ++i){
            for(int j = 0; j < r4; ++j){
                for(int k = 0; k < 3; ++k){
                    ifs1>>tmp;
                    // cout<<tmp<<"\n";
                    q_img[(r3-1-i)*r4*3 +j*3 +k] = tmp;
                    // cout<<img[(x-1-i)*y*3 +j*3 +k];
                }
            }
        }
        // cout<<"HELLO"<<endl;
        ifs1.close();
    }
    dim2 = {r3,r4};
    // dim1 = read_img(data_img_file, data_img);
    // dim2 = read_img(q_img_file, q_img);

    float q_avg = 0;
    for(int x=0;x<dim2.first;++x){
        for(int y=0;y<dim2.second;++y){
            q_avg+=((float)(q_img[x*dim2.second*3+y*3+0]+q_img[x*dim2.second*3+y*3+1]+q_img[x*dim2.second*3+y*3+2]));
        }
    }

    q_avg = q_avg/((float)(3*dim2.first*dim2.second));
    cout<<"q_avg "<<q_avg<<endl;

    // float t_avg = 0;
    // for(int x=840;x<840+dim2.first;++x){
    //     for(int y=900;y<900+dim2.second;++y){
    //         t_avg+=((float)(data_img[x*dim1.second*3+y*3+0]+data_img[x*dim1.second*3+y*3+1]+data_img[x*dim1.second*3+y*3+2]));
    //     }
    // }

    // t_avg = t_avg/((float)(3*dim2.first*dim2.second));
    // cout<<"t_avg "<<t_avg<<endl;

    float* rmsd;
    // rmsd = new float[dim1.first*dim1.second*3];
    hipMallocManaged(&rmsd, dim1.first*dim1.second*3*sizeof(float));
    for(int i=0;i<dim1.first*dim1.second*3;++i){
        rmsd[i]=-1;
    }

    int m = dim1.second;
    int n1 = dim1.first;
    int b = dim2.second;
    int a = dim2.first;
    // cout<<n1<<" "<<m<<" "<<a<<" "<<b<<"\n";
    dim3 gd(n1,m);
    main_func<<<gd, 1>>>(data_img, n1, m, q_img, a, b, rmsd, q_avg, th1, th2);

    hipDeviceSynchronize();

    priority_queue<pair<float, vector<int>>> pq;
    int c=0;
    for(int i=0;i<dim1.first;++i){
        for(int j=0;j<dim1.second;++j){
            for(int k=0;k<3;++k){
                float tmp = rmsd[i*m*3+j*3+k];
                if(tmp!=-1){
                    pq.push({-tmp,{i,j,k}});
                    c++;
                    // cout<<i<<" "<<j<<" "<<k<<" "<<c<<"\n";
                }
            }
        }
    }

    cout<<c<<"\n";

    ofstream ofs("output.txt");

    if(ofs.is_open()){
        for(int i=0; i<min(n,c);++i){
            auto tmp = pq.top();
            pq.pop();
            // cout<<-tmp.first<<"\n";
            ofs<<tmp.second[0]<<" "<<tmp.second[1]<<" ";
            if(tmp.second[2]==0){
                ofs<<0<<"\n";
            } 
            else if(tmp.second[2]==1){
                ofs<<45<<"\n";
            }
            else{
                ofs<<-45<<"\n";
            }
        }
        ofs.close();
    }

    hipFree(data_img);
    hipFree(q_img);
    hipFree(rmsd);

    return 0;
}